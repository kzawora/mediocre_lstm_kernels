
#include <hip/hip_runtime.h>
__device__ __forceinline__ double sigmoid (double a) { return 1.0 / (1.0 + exp (-a)); }
__device__ __forceinline__ int idx_2d(int x, int y, int width) { return x*width+y; }
__device__ __forceinline__ float dp4a_fp32(float4 a, float4 b) { return a.x * b.x + a.y * b.y + a.z * b.z + a.w * b.w; }
__device__ __forceinline__ void printf4(float4 a) {  printf("%f %f %f %f\n", a.x, a.y, a.z, a.w); }

__global__ void lstm_gemm(float *input,
                          float *initial_hiddens,
                          float *weights,
                          float *bias,
                          float *out_gates,
                          int M, int K, int N,
                          int input_size, int hidden_size, int timestep) {
    int m = threadIdx.x + blockIdx.x * blockDim.x;
    int n = threadIdx.y + blockIdx.y * blockDim.y;
   // if (!(m == 0 && n == 0)) return;
    int c_wr_idx = idx_2d(m,n,N);
    float acc = 0.;
    float4* input4 = reinterpret_cast<float4*>(input);
    float4* hidden4 = reinterpret_cast<float4*>(initial_hiddens);
    float4* weights4 = reinterpret_cast<float4*>(weights);
    for (int k = 0; k < K/4; k++) {
        float4 a_matrix_elem = k < input_size/4 ? input4[idx_2d(m,k,input_size/4) + M*input_size/4*timestep ]
                                                : hidden4[idx_2d(m,k-input_size/4,hidden_size/4) + M*hidden_size/4*timestep];
        float4 b_matrix_elem = weights4[idx_2d(n,k,K/4)];
        acc += dp4a_fp32(a_matrix_elem, b_matrix_elem);
    }
    out_gates[c_wr_idx] = acc + bias[n];
}

__global__ void lstm_eltwise(float* inout_cell,
                             float *out_gates,
                             float*hidden_out,
                             int hidden_size,
                             int batch_size,
                             int timestep) {

    int m = threadIdx.x + blockIdx.x * blockDim.x;
    int n = threadIdx.y + blockIdx.y * blockDim.y;
    float4* out_gates4 = reinterpret_cast<float4*>(out_gates);
    float4 ifgo = out_gates4[idx_2d(m,n,hidden_size)];
    float i = ifgo.x;
    float f = ifgo.y;
    float g = ifgo.z;
    float o = ifgo.w;

    float cell = sigmoid(f) * inout_cell[idx_2d(m, n, hidden_size) + batch_size * hidden_size * timestep]  + sigmoid(i) * tanh(g);
    float hidden = sigmoid(o) * tanh(cell);

    int hidden_wr_timestep_offset = batch_size * hidden_size * (timestep+1);
    int hidden_wr_idx = idx_2d(m, n, hidden_size) + hidden_wr_timestep_offset;
    int cell_wr_idx = idx_2d(m, n, hidden_size) + hidden_wr_timestep_offset;
    hidden_out[hidden_wr_idx] = hidden;
    inout_cell[cell_wr_idx] = cell;
}

