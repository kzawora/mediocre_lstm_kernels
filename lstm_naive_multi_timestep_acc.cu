
#include <hip/hip_runtime.h>
__device__ __forceinline__ double sigmoid (double a) { return 1.0 / (1.0 + exp (-a)); }
__device__ __forceinline__ int idx_2d(int x, int y, int width) { return x*width+y; }
__global__ void lstm_gemm(float *input,
                          float *initial_hiddens,
                          float *weights,
                          float *bias,
                          float *out_gates,
                          int M, int K, int N,
                          int input_size, int hidden_size, int timestep) {
    int m = threadIdx.x + blockIdx.x * blockDim.x;
    int n = threadIdx.y + blockIdx.y * blockDim.y;
    int c_wr_idx = idx_2d(m,n,N);
    float acc = 0.;
    for (int k = 0; k < K; k++) {
        int b_rd_idx = idx_2d(k,n,N);
        float a_matrix_elem = k < input_size ? input[idx_2d(m,k,input_size) + M*input_size*timestep ]
                                             : initial_hiddens[idx_2d(m,k-input_size,hidden_size) + M*hidden_size*timestep];
        acc += a_matrix_elem * weights[b_rd_idx];
    }
    out_gates[c_wr_idx] = acc + bias[n];
}

__global__ void lstm_eltwise(float* inout_cell,
                             float *out_gates,
                             float*hidden_out,
                             int hidden_size,
                             int batch_size,
                             int timestep) {

    int m = threadIdx.x + blockIdx.x * blockDim.x;
    int n = threadIdx.y + blockIdx.y * blockDim.y;

    int i_idx = idx_2d(m,0*hidden_size+n,4*hidden_size);
    int f_idx = idx_2d(m,1*hidden_size+n,4*hidden_size);
    int g_idx = idx_2d(m,2*hidden_size+n,4*hidden_size);
    int o_idx = idx_2d(m,3*hidden_size+n,4*hidden_size);

    float i = out_gates[i_idx];
    float f = out_gates[f_idx];
    float g = out_gates[g_idx];
    float o = out_gates[o_idx];

    float cell = sigmoid(f) * inout_cell[idx_2d(m, n, hidden_size) + batch_size * hidden_size * timestep]  + sigmoid(i) * tanh(g);
    float hidden = sigmoid(o) * tanh(cell);

    int hidden_wr_timestep_offset = batch_size * hidden_size * (timestep+1);
    int hidden_wr_idx = idx_2d(m, n, hidden_size) + hidden_wr_timestep_offset;
    int cell_wr_idx = idx_2d(m, n, hidden_size) + hidden_wr_timestep_offset;
    hidden_out[hidden_wr_idx] = hidden;
    inout_cell[cell_wr_idx] = cell;
}

